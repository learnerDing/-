#include "hip/hip_runtime.h"
#include "src/kernels/build_casual_mask.h"
// XuLin-1017: 此算子仅使用在context decoder阶段，用于遮盖掉seq当前位置之后的信息，防止模型使用未来的信息
// 而self decoder是一个自回归模型，本就没有未来的信息
// mask shape =  [bs, max_q_len, max_k_len] 因为Q矩阵大小为n*hidden_size，Kt矩阵shape:hidden_size*m，所以Q*Kt矩阵为n*m维度
template<typename T>
__global__ void BuildCausalMasksConsideringContextPastKV(T* mask,//mask张量开头的地址
                                                const int* q_lens,  //input lens,q数组， shape=[batch size]   batchsize个句子
                                                const int* k_lens,  //context lens, shape=[batch size]
                                                int max_q_len,  // max(q_lens)
                                                int max_k_len){ // max(k_lens)
    int tid = threadIdx.x;
    // XuLin-1017: 核函数共分配了bs个block，可以方便得通过block id来访问q_lens和k_lens数组中的值
    // 一个block负责处理一个bs大小中的数
    int qlen = q_lens[blockIdx.x];
    int klen = k_lens[blockIdx.x];
    // 偏移一个bs大小的空间
    // 即blockIdx.x==0时，指向mask数组的开头；blockIdx.x==1时，指向mask数组偏移了max_q_len * max_k_len大小后的位置
    mask += blockIdx.x * max_q_len * max_k_len;
    // offset用于表示每个bs内部的偏移
    int offset = threadIdx.x;
    // note: this judgement confirms we dont exceed data boundry
    while (offset < max_q_len * max_k_len){//循环生成矩阵shape为max_q_len行，max_k_len列
        // 分别求出行号q和列号k
        int q = offset / max_k_len;//该线程（数据）在矩阵的第q行
        int k = offset % max_k_len;//该线程（数据）在矩阵的第k列
        // 此处与视频中的代码不同，k考虑了多轮对话的上下文序列，但设置mask时 k >= klen - qlen 将旧序列一并遮去了
        // 下图为支持多轮对话的mask，第二轮对话中有些token需要掩盖，但是第一轮对话中的所有token都是已知的，所有左边矩阵全为一
        // 1 1 1 | 1 -inf -inf
        // 1 1 1 | 1   1  -inf
        // 1 1 1 | 1   1    1
        // 下图为不支持多轮对话的mask，上一轮矩阵数据全部被屏蔽
        // -inf -inf -inf | 1 -inf -inf
        // -inf -inf -inf | 1   1  -inf
        // -inf -inf -inf | 1   1    1
        // "|"符号前表示旧的对话序列，符号后表示当前轮的对话序列
        bool is_one = q < qlen && k < klen && k <= q + (klen - qlen) && k >= klen - qlen;//判断该矩阵哪些位置应该填1
        mask[offset] = static_cast<T>(is_one);

        // 保证遍历完一个bs中所有的空间
        offset += blockDim.x;
    }
}

template<typename T>
void launchBuildCausalMasks(TensorWrapper<T>* mask, 
                            TensorWrapper<int>* q_lens, 
                            TensorWrapper<int>* k_lens)
{
    int batch_size = mask->shape[0];
    int max_q_len = mask->shape[1];
    int max_k_len = mask->shape[2];
    // XuLin-1017: 此处的max_q_len和max_k_len是经过统计后得出的外部输入
    BuildCausalMasksConsideringContextPastKV<T><<<batch_size, 256>>>(mask->data, q_lens->data, k_lens->data, max_q_len, max_k_len);
}

template void launchBuildCausalMasks(TensorWrapper<float>* mask, 
                            TensorWrapper<int>* q_lens, 
                            TensorWrapper<int>* k_lens);

template void launchBuildCausalMasks(TensorWrapper<half>* mask, 
                            TensorWrapper<int>* q_lens, 
                            TensorWrapper<int>* k_lens);
