#include "hip/hip_runtime.h"
#include "src/kernels/cal_paddingoffset.h"
// shape:
    //seq_lengths:[batch size]输入句子长度
    //cum_seqlens:[batch size + 1],first ele is 0  累计的句子长度
    //padding_offset:[batch size * max q len]
// note: the point is to calc padding offset and cum offset
// TODO: we first use serial algo, then can enhance to CUDA scan algo

// zhaziqwe : 并行版本的前缀和实现推荐文章 ：https://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/scan/doc/scan.pdf

// 例子 bs = 3, seqlen = [3,2,5], max_seq_len = 5
// 1 1 1 0 0
// 1 1 0 0 0
// 1 1 1 1 1
// paddingoffset 为
// 0 0 0 0 0
// 2 2 2 2 2
// 5 5 5 5 5
//total_seqlens为
//1 2 3 0 0 
//4 5 0 0 0
//6 7 8 9 10
//串行kernel

__global__ void CalPaddingoffset(int*         padding_offset, //每个token都有一个
                                int*         cum_seqlens,//输出累计的句子长度,每行输出一个
                                const int*   input_lengths, //actual input lens  输入
                                const int    batch_size,
                                const int    max_q_len) {
    int ind = 0;
    int cum_offset = 0;
    int total_seqlen = 0;
    // 遍历每一个批次
    for(int b = 0; b < batch_size; b++) {
        // 获取到每个句子的长度
        int seqlen = input_lengths[b];
        // 累计的句子长度
        cum_seqlens[b] = total_seqlen;
        // 遍历一个句子里的所有token位置
        // each token in one seq has same cum offset
        for (int i = 0; i < seqlen; i++) {//遍历单个句子
            // ind是对于每个token的索引，每个token都有一个paddingoffset
            padding_offset[ind] = cum_offset;
            ind++;
        }
        // 获取累计的 padding offset 和 总共的句子长度
        cum_offset += max_q_len - seqlen;
        total_seqlen += seqlen;
    }
    // 注意 cum_seqlens 的形状，添加最后一个累计句子长度（总长度）
    cum_seqlens[batch_size] = total_seqlen;
}

// zhaziqwe： 这个函数的目的是为了在attention之后，可以方便的移除padding。
// padding操作和 seq len 维度相关，因此相关操作需要在不涉及这一维度的计算后添加。  
void launchCalPaddingoffset(TensorWrapper<int>* padding_offset, //偏移之后的数据
                            TensorWrapper<int>* cum_seqlens,//累计的句子长度
                            TensorWrapper<int>* input_lengths)//actual input lens输入长度
{
    const int batch_size = padding_offset->shape[0];                            
    const int max_q_len = padding_offset->shape[1]; 
    LLM_CHECK_WITH_INFO(batch_size == input_lengths->shape[0], "input lenghts numbers should equal to padding offset bs dim!") ;                        
    LLM_CHECK_WITH_INFO(batch_size == cum_seqlens->shape[0] - 1, "cum seqlen numbers should equal to padding offset bs dim + 1!") ;                        
    CalPaddingoffset<<<1, 1>>>( 
        padding_offset->data, cum_seqlens->data, input_lengths->data, batch_size, max_q_len
    );
}