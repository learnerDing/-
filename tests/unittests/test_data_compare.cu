#include "hip/hip_runtime.h"
#include <algorithm>   // std::fill_n
#include <iostream>    // snprintf
#include <math.h>      // expf, log
#include <stdlib.h>    // rand
#include <string>      // std::string
#include <vector>      // std::vector
#include <stdio.h>
#include <fstream>
#include "src/utils/macro.h"
#include "src/utils/debug_utils.h"
// (RussWong)note:
// this test is for debug, to compare intermediate tensor and HF intermediate tensor
// and the intermediate tensor will be saved in file when you compile the proj by `cmake .. -DSAVE_DATA=ON && make -j8`
// before run, you should change the path to your local right dir
// `./debug` to compare 

std::vector<float> loadWeightFromBinHelper(std::vector<size_t> shape, std::string filename)
{
    size_t dim0 = 1, dim1 = 1;
    if (shape.size() > 2) {
        dim0 = shape[0] * shape[1];
        dim1 = shape[2];
    }

    if (shape.size() == 2) {
        dim0 = shape[0];
        dim1 = shape[1];
    }
    size_t size = dim0 * dim1;
    if (size == 0) {
        std::cout << "shape is zero, skip loading weight from file: " << filename << std::endl;
        return std::vector<float>();
    }

    std::vector<float> host_array(size);
    std::ifstream  in(filename, std::ios::in | std::ios::binary);
    if (!in.is_open()) {
        std::cout << "file" << filename << "cannot be opened, loading model fails!" << std::endl;
        return std::vector<float>();
    }

    size_t loaded_data_size = sizeof(float) * size;
    in.seekg(0, in.end);
    in.seekg(0, in.beg);

    std::cout << "Read " << std::to_string(loaded_data_size) << " bytes from " << filename << std::endl;
    in.read((char*)host_array.data(), loaded_data_size);

    size_t in_get_size = in.gcount();
    if (in_get_size != loaded_data_size) {
        return std::vector<float>();
    }
    in.close();
    // If we succeed, return an array with values.
    return host_array;
}
void internalFunc(float* ptr, std::vector<size_t> shape, std::string filename) {
    std::vector<float> host_array = loadWeightFromBinHelper(shape, filename);
    if (host_array.empty()) {
        std::cout << "[warning] data from file is empty!!" << "\n";
        return;
    }
    // copy host_array to our defined ptr
    memcpy(ptr, host_array.data(), host_array.size());
    return;
}
void loadWeights(float* ptr1, std::string weight_path, int shape0, int shape1) // weighttype参数比较多余
{
    // load attn output
    internalFunc(ptr1, {(size_t)shape0, (size_t)shape1}, weight_path);

}
void loadWeights_trans(float* ptr1, std::string weight_path, int shape0, int shape1) // weighttype参数比较多余
{
    // load attn output
    internalFunc(ptr1, {(size_t)shape0, (size_t)shape1}, weight_path);

}

bool CheckResult(float* CPUoutput, float* GPUoutput, int in_size) {
    for(int i = 0; i < in_size; i++) {
	if(fabs(CPUoutput[i] - GPUoutput[i]) > 1e-6){
	    printf("the %dth res is wrong, onellm = %f, trans = %f\n", i, CPUoutput[i], GPUoutput[i]);
    	}
    }
    return true;
}
// 1.for example: the path of two data files is below, and you should replace L101&L102 with the two
// /home/data/trans/q_buf_after_rope_trans.bin
// /home/data/onellm/q_buf_after_rope.bin
// 2.And you should change the L93&L94 to the right data size according to your tensor shape of the data file
int main(int argc, char *argv[]) {
    int shape0 = 1; // TO MODIFY before run
    int shape1 = 4096; // TO MODIFY before run
    
    int in_size = shape0 * shape1;

    float* d_in = (float*) malloc(sizeof(float) * in_size);
    float* d_in_trans = (float*) malloc(sizeof(float) * in_size);

    loadWeights(d_in, "/home/data/onellm/0_self_decoder_qk_v_after_bmm.bin", shape0, shape1); // TO MODIFY
    loadWeights_trans(d_in_trans, "/home/data/trans/self_decoder_qk_v_buf_after_bmm_trans.bin", shape0, shape1); // TO MODIFY
    std::cout << "====intermediate tensor comparison result====" << "\n";
    CheckResult(d_in, d_in_trans, shape0 * shape1);

    free(d_in);
    free(d_in_trans);

}
